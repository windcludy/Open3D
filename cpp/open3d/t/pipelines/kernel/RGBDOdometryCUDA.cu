#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018-2021 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>

#include "open3d/core/CUDAUtils.h"
#include "open3d/core/Dispatch.h"
#include "open3d/core/ParallelFor.h"
#include "open3d/core/Tensor.h"
#include "open3d/t/geometry/kernel/GeometryIndexer.h"
#include "open3d/t/geometry/kernel/GeometryMacros.h"
#include "open3d/t/pipelines/kernel/RGBDOdometryImpl.h"
#include "open3d/t/pipelines/kernel/RGBDOdometryJacobianImpl.h"
#include "open3d/t/pipelines/kernel/TransformationConverter.h"
#include "open3d/utility/MiniVec.h"

namespace open3d {
namespace t {
namespace pipelines {
namespace kernel {
namespace odometry {

const int kBlockSize = 256;
const int kReduceDim = 29;  // 21 (JtJ) + 6 (Jtr) + 1 (inlier) + 1 (r)
typedef utility::MiniVec<float, kReduceDim> ReduceVec;
typedef hipcub::BlockReduce<ReduceVec, kBlockSize> BlockReduce;

__global__ void ComputeOdometryResultPointToPlaneCUDAKernel(
        NDArrayIndexer source_vertex_indexer,
        NDArrayIndexer target_vertex_indexer,
        NDArrayIndexer target_normal_indexer,
        TransformIndexer ti,
        float* global_sum,
        int rows,
        int cols,
        const float depth_outlier_trunc,
        const float depth_huber_delta) {
    __shared__ typename BlockReduce::TempStorage temp_storage;

    const int workload = threadIdx.x + blockIdx.x * blockDim.x;
    int y = workload / cols;
    int x = workload % cols;
    const int tid = threadIdx.x;

    ReduceVec local_sum;
    if (workload < rows * cols) {
        float J[6] = {0};
        float r = 0;
        bool valid = GetJacobianPointToPlane(
                x, y, depth_outlier_trunc, source_vertex_indexer,
                target_vertex_indexer, target_normal_indexer, ti, J, r);

        float d_huber = HuberDeriv(r, depth_huber_delta);
        float r_huber = HuberLoss(r, depth_huber_delta);

        // Dump J, r into JtJ and Jtr
        int offset = 0;
        for (int i = 0; i < 6; ++i) {
            for (int j = 0; j <= i; ++j) {
                local_sum[offset++] = valid ? J[i] * J[j] : 0;
            }
        }
        for (int i = 0; i < 6; ++i) {
            local_sum[offset++] = valid ? J[i] * d_huber : 0;
        }
        local_sum[offset++] = valid ? r_huber : 0;
        local_sum[offset++] = valid;
    }

    auto result = BlockReduce(temp_storage).Sum(local_sum);
    if (tid == 0) {
#pragma unroll
        for (int i = 0; i < kReduceDim; ++i) {
            atomicAdd(&global_sum[i], result[i]);
        }
    }
}

void ComputeOdometryResultPointToPlaneCUDA(
        const core::Tensor& source_vertex_map,
        const core::Tensor& target_vertex_map,
        const core::Tensor& target_normal_map,
        const core::Tensor& intrinsics,
        const core::Tensor& init_source_to_target,
        core::Tensor& delta,
        float& inlier_residual,
        int& inlier_count,
        const float depth_outlier_trunc,
        const float depth_huber_delta) {
    NDArrayIndexer source_vertex_indexer(source_vertex_map, 2);
    NDArrayIndexer target_vertex_indexer(target_vertex_map, 2);
    NDArrayIndexer target_normal_indexer(target_normal_map, 2);

    core::Device device = source_vertex_map.GetDevice();

    core::Tensor trans = init_source_to_target;
    TransformIndexer ti(intrinsics, trans);

    const int64_t rows = source_vertex_indexer.GetShape(0);
    const int64_t cols = source_vertex_indexer.GetShape(1);

    core::Tensor global_sum =
            core::Tensor::Zeros({kReduceDim}, core::Float32, device);
    float* global_sum_ptr = global_sum.GetDataPtr<float>();

    const dim3 blocks((rows * cols + kBlockSize - 1) / kBlockSize);
    const dim3 threads(kBlockSize);
    ComputeOdometryResultPointToPlaneCUDAKernel<<<blocks, threads, 0,
                                                  core::cuda::GetStream()>>>(
            source_vertex_indexer, target_vertex_indexer, target_normal_indexer,
            ti, global_sum_ptr, rows, cols, depth_outlier_trunc,
            depth_huber_delta);
    core::cuda::Synchronize();
    DecodeAndSolve6x6(global_sum, delta, inlier_residual, inlier_count);
}

__global__ void ComputeOdometryResultIntensityCUDAKernel(
        NDArrayIndexer source_depth_indexer,
        NDArrayIndexer target_depth_indexer,
        NDArrayIndexer source_intensity_indexer,
        NDArrayIndexer target_intensity_indexer,
        NDArrayIndexer target_intensity_dx_indexer,
        NDArrayIndexer target_intensity_dy_indexer,
        NDArrayIndexer source_vertex_indexer,
        TransformIndexer ti,
        float* global_sum,
        int rows,
        int cols,
        const float depth_outlier_trunc,
        const float intensity_huber_delta) {
    __shared__ typename BlockReduce::TempStorage temp_storage;

    const int workload = threadIdx.x + blockIdx.x * blockDim.x;
    int y = workload / cols;
    int x = workload % cols;
    const int tid = threadIdx.x;

    ReduceVec local_sum;
    if (workload < rows * cols) {
        float J[6] = {0};
        float r = 0;
        bool valid = GetJacobianIntensity(
                x, y, depth_outlier_trunc, source_depth_indexer,
                target_depth_indexer, source_intensity_indexer,
                target_intensity_indexer, target_intensity_dx_indexer,
                target_intensity_dy_indexer, source_vertex_indexer, ti, J, r);

        float d_huber = HuberDeriv(r, intensity_huber_delta);
        float r_huber = HuberLoss(r, intensity_huber_delta);

        // Dump J, r into JtJ and Jtr
        int offset = 0;
        for (int i = 0; i < 6; ++i) {
            for (int j = 0; j <= i; ++j) {
                local_sum[offset++] = J[i] * J[j];
            }
        }
        for (int i = 0; i < 6; ++i) {
            local_sum[offset++] = J[i] * HuberDeriv(r, intensity_huber_delta);
        }
        local_sum[offset++] = HuberLoss(r, intensity_huber_delta);
        local_sum[offset++] = valid;
    }

    auto result = BlockReduce(temp_storage).Sum(local_sum);
    if (tid == 0) {
#pragma unroll
        for (int i = 0; i < kReduceDim; ++i) {
            atomicAdd(&global_sum[i], result[i]);
        }
    }
}

void ComputeOdometryResultIntensityCUDA(
        const core::Tensor& source_depth,
        const core::Tensor& target_depth,
        const core::Tensor& source_intensity,
        const core::Tensor& target_intensity,
        const core::Tensor& target_intensity_dx,
        const core::Tensor& target_intensity_dy,
        const core::Tensor& source_vertex_map,
        const core::Tensor& intrinsics,
        const core::Tensor& init_source_to_target,
        core::Tensor& delta,
        float& inlier_residual,
        int& inlier_count,
        const float depth_outlier_trunc,
        const float intensity_huber_delta) {
    NDArrayIndexer source_depth_indexer(source_depth, 2);
    NDArrayIndexer target_depth_indexer(target_depth, 2);

    NDArrayIndexer source_intensity_indexer(source_intensity, 2);
    NDArrayIndexer target_intensity_indexer(target_intensity, 2);

    NDArrayIndexer target_intensity_dx_indexer(target_intensity_dx, 2);
    NDArrayIndexer target_intensity_dy_indexer(target_intensity_dy, 2);

    NDArrayIndexer source_vertex_indexer(source_vertex_map, 2);

    core::Device device = source_vertex_map.GetDevice();
    core::Tensor trans = init_source_to_target;
    t::geometry::kernel::TransformIndexer ti(intrinsics, trans);

    const int64_t rows = source_vertex_indexer.GetShape(0);
    const int64_t cols = source_vertex_indexer.GetShape(1);

    core::Tensor global_sum =
            core::Tensor::Zeros({kReduceDim}, core::Float32, device);
    float* global_sum_ptr = global_sum.GetDataPtr<float>();

    const dim3 blocks((cols * rows + kBlockSize - 1) / kBlockSize);
    const dim3 threads(kBlockSize);
    ComputeOdometryResultIntensityCUDAKernel<<<blocks, threads, 0,
                                               core::cuda::GetStream()>>>(
            source_depth_indexer, target_depth_indexer,
            source_intensity_indexer, target_intensity_indexer,
            target_intensity_dx_indexer, target_intensity_dy_indexer,
            source_vertex_indexer, ti, global_sum_ptr, rows, cols,
            depth_outlier_trunc, intensity_huber_delta);
    core::cuda::Synchronize();
    DecodeAndSolve6x6(global_sum, delta, inlier_residual, inlier_count);
}

__global__ void ComputeOdometryResultHybridCUDAKernel(
        NDArrayIndexer source_depth_indexer,
        NDArrayIndexer target_depth_indexer,
        NDArrayIndexer source_intensity_indexer,
        NDArrayIndexer target_intensity_indexer,
        NDArrayIndexer target_depth_dx_indexer,
        NDArrayIndexer target_depth_dy_indexer,
        NDArrayIndexer target_intensity_dx_indexer,
        NDArrayIndexer target_intensity_dy_indexer,
        NDArrayIndexer source_vertex_indexer,
        TransformIndexer ti,
        float* global_sum,
        int rows,
        int cols,
        const float depth_outlier_trunc,
        const float depth_huber_delta,
        const float intensity_huber_delta) {
    __shared__ typename BlockReduce::TempStorage temp_storage;

    const int workload = threadIdx.x + blockIdx.x * blockDim.x;
    int y = workload / cols;
    int x = workload % cols;
    const int tid = threadIdx.x;

    ReduceVec local_sum;
    if (workload < rows * cols) {
        float J_I[6] = {0}, J_D[6] = {0};
        float r_I = 0, r_D = 0;
        bool valid = GetJacobianHybrid(
                x, y, depth_outlier_trunc, source_depth_indexer,
                target_depth_indexer, source_intensity_indexer,
                target_intensity_indexer, target_depth_dx_indexer,
                target_depth_dy_indexer, target_intensity_dx_indexer,
                target_intensity_dy_indexer, source_vertex_indexer, ti, J_I,
                J_D, r_I, r_D);

        float d_huber_D = HuberDeriv(r_D, depth_huber_delta);
        float d_huber_I = HuberDeriv(r_I, intensity_huber_delta);

        float r_huber_D = HuberLoss(r_D, depth_huber_delta);
        float r_huber_I = HuberLoss(r_I, intensity_huber_delta);

        // Dump J, r into JtJ and Jtr
        int offset = 0;
        for (int i = 0; i < 6; ++i) {
            for (int j = 0; j <= i; ++j) {
                local_sum[offset++] = J_I[i] * J_I[j] + J_D[i] * J_D[j];
            }
        }
        for (int i = 0; i < 6; ++i) {
            local_sum[offset++] = J_I[i] * d_huber_I + J_D[i] * d_huber_D;
        }
        local_sum[offset++] = r_huber_D + r_huber_I;
        local_sum[offset++] = valid;
    }

    auto result = BlockReduce(temp_storage).Sum(local_sum);
    if (tid == 0) {
#pragma unroll
        for (int i = 0; i < kReduceDim; ++i) {
            atomicAdd(&global_sum[i], result[i]);
        }
    }
}

void ComputeOdometryResultHybridCUDA(const core::Tensor& source_depth,
                                     const core::Tensor& target_depth,
                                     const core::Tensor& source_intensity,
                                     const core::Tensor& target_intensity,
                                     const core::Tensor& target_depth_dx,
                                     const core::Tensor& target_depth_dy,
                                     const core::Tensor& target_intensity_dx,
                                     const core::Tensor& target_intensity_dy,
                                     const core::Tensor& source_vertex_map,
                                     const core::Tensor& intrinsics,
                                     const core::Tensor& init_source_to_target,
                                     core::Tensor& delta,
                                     float& inlier_residual,
                                     int& inlier_count,
                                     const float depth_outlier_trunc,
                                     const float depth_huber_delta,
                                     const float intensity_huber_delta) {
    NDArrayIndexer source_depth_indexer(source_depth, 2);
    NDArrayIndexer target_depth_indexer(target_depth, 2);

    NDArrayIndexer source_intensity_indexer(source_intensity, 2);
    NDArrayIndexer target_intensity_indexer(target_intensity, 2);

    NDArrayIndexer target_depth_dx_indexer(target_depth_dx, 2);
    NDArrayIndexer target_depth_dy_indexer(target_depth_dy, 2);
    NDArrayIndexer target_intensity_dx_indexer(target_intensity_dx, 2);
    NDArrayIndexer target_intensity_dy_indexer(target_intensity_dy, 2);

    NDArrayIndexer source_vertex_indexer(source_vertex_map, 2);

    core::Device device = source_vertex_map.GetDevice();
    core::Tensor trans = init_source_to_target;
    t::geometry::kernel::TransformIndexer ti(intrinsics, trans);

    const int64_t rows = source_vertex_indexer.GetShape(0);
    const int64_t cols = source_vertex_indexer.GetShape(1);

    core::Tensor global_sum =
            core::Tensor::Zeros({kReduceDim}, core::Float32, device);
    float* global_sum_ptr = global_sum.GetDataPtr<float>();

    const dim3 blocks((cols * rows + kBlockSize - 1) / kBlockSize);
    const dim3 threads(kBlockSize);
    ComputeOdometryResultHybridCUDAKernel<<<blocks, threads, 0,
                                            core::cuda::GetStream()>>>(
            source_depth_indexer, target_depth_indexer,
            source_intensity_indexer, target_intensity_indexer,
            target_depth_dx_indexer, target_depth_dy_indexer,
            target_intensity_dx_indexer, target_intensity_dy_indexer,
            source_vertex_indexer, ti, global_sum_ptr, rows, cols,
            depth_outlier_trunc, depth_huber_delta, intensity_huber_delta);
    core::cuda::Synchronize();
    DecodeAndSolve6x6(global_sum, delta, inlier_residual, inlier_count);
}

}  // namespace odometry
}  // namespace kernel
}  // namespace pipelines
}  // namespace t
}  // namespace open3d
