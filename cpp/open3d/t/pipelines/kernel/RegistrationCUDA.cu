#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018-2021 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>

#include "open3d/core/CUDAUtils.h"
#include "open3d/core/ParallelFor.h"
#include "open3d/core/Tensor.h"
#include "open3d/t/pipelines/kernel/RegistrationImpl.h"
#include "open3d/t/pipelines/kernel/TransformationConverter.h"
#include "open3d/t/pipelines/registration/RobustKernel.h"
#include "open3d/t/pipelines/registration/RobustKernelImpl.h"
#include "open3d/utility/MiniVec.h"

namespace open3d {
namespace t {
namespace pipelines {
namespace kernel {

const int kThread1DUnit = 256;
const int kReduceDim = 29;  // 21 (JtJ) + 6 (Jtr) + 1 (inlier) + 1 (r)

template <typename scalar_t, typename func_t>
__global__ void ComputePosePointToPlaneKernelCUDA(
        const scalar_t *source_points_ptr,
        const scalar_t *target_points_ptr,
        const scalar_t *target_normals_ptr,
        const int64_t *correspondence_indices,
        const int n,
        scalar_t *global_sum,
        func_t GetWeightFromRobustKernel) {
    typedef utility::MiniVec<scalar_t, kReduceDim> ReduceVec;
    // Create shared memory.
    typedef hipcub::BlockReduce<ReduceVec, kThread1DUnit> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    ReduceVec local_sum(static_cast<scalar_t>(0));

    const int workload_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (workload_idx < n) {
        scalar_t J_ij[6] = {0};
        scalar_t r = 0;
        const bool valid = GetJacobianPointToPlane<scalar_t>(
                workload_idx, source_points_ptr, target_points_ptr,
                target_normals_ptr, correspondence_indices, J_ij, r);

        if (valid) {
            const scalar_t w = GetWeightFromRobustKernel(r);

            // Dump J, r into JtJ and Jtr
            int i = 0;
            for (int j = 0; j < 6; ++j) {
                for (int k = 0; k <= j; ++k) {
                    local_sum[i] += J_ij[j] * w * J_ij[k];
                    ++i;
                }
                local_sum[21 + j] += J_ij[j] * w * r;
            }
            local_sum[27] += r;
            local_sum[28] += 1;
        }
    }

    // Reduction.
    auto result = BlockReduce(temp_storage).Sum(local_sum);

    // Add result to global_sum.
    if (threadIdx.x == 0) {
#pragma unroll
        for (int i = 0; i < kReduceDim; ++i) {
            atomicAdd(&global_sum[i], result[i]);
        }
    }
}

void ComputePosePointToPlaneCUDA(const core::Tensor &source_points,
                                 const core::Tensor &target_points,
                                 const core::Tensor &target_normals,
                                 const core::Tensor &correspondence_indices,
                                 core::Tensor &pose,
                                 float &residual,
                                 int &inlier_count,
                                 const core::Dtype &dtype,
                                 const core::Device &device,
                                 const registration::RobustKernel &kernel) {
    int n = source_points.GetLength();

    core::Tensor global_sum = core::Tensor::Zeros({29}, dtype, device);
    const dim3 blocks((n + kThread1DUnit - 1) / kThread1DUnit);
    const dim3 threads(kThread1DUnit);

    DISPATCH_FLOAT_DTYPE_TO_TEMPLATE(dtype, [&]() {
        scalar_t *global_sum_ptr = global_sum.GetDataPtr<scalar_t>();

        DISPATCH_ROBUST_KERNEL_FUNCTION(
                kernel.type_, scalar_t, kernel.scaling_parameter_,
                kernel.shape_parameter_, [&]() {
                    ComputePosePointToPlaneKernelCUDA<<<
                            blocks, threads, 0, core::cuda::GetStream()>>>(
                            source_points.GetDataPtr<scalar_t>(),
                            target_points.GetDataPtr<scalar_t>(),
                            target_normals.GetDataPtr<scalar_t>(),
                            correspondence_indices.GetDataPtr<int64_t>(), n,
                            global_sum_ptr, GetWeightFromRobustKernel);
                });
    });

    core::cuda::Synchronize();

    DecodeAndSolve6x6(global_sum, pose, residual, inlier_count);
}

template <typename scalar_t, typename funct_t>
__global__ void ComputePoseColoredICPKernelCUDA(
        const scalar_t *source_points_ptr,
        const scalar_t *source_colors_ptr,
        const scalar_t *target_points_ptr,
        const scalar_t *target_normals_ptr,
        const scalar_t *target_colors_ptr,
        const scalar_t *target_color_gradients_ptr,
        const int64_t *correspondence_indices,
        const scalar_t sqrt_lambda_geometric,
        const scalar_t sqrt_lambda_photometric,
        const int n,
        scalar_t *global_sum,
        funct_t GetWeightFromRobustKernel) {
    typedef utility::MiniVec<scalar_t, kReduceDim> ReduceVec;
    // Create shared memory.
    typedef hipcub::BlockReduce<ReduceVec, kThread1DUnit> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    ReduceVec local_sum(static_cast<scalar_t>(0));

    const int workload_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (workload_idx < n) {
        scalar_t J_G[6] = {0}, J_I[6] = {0};
        scalar_t r_G = 0, r_I = 0;

        const bool valid = GetJacobianColoredICP<scalar_t>(
                workload_idx, source_points_ptr, source_colors_ptr,
                target_points_ptr, target_normals_ptr, target_colors_ptr,
                target_color_gradients_ptr, correspondence_indices,
                sqrt_lambda_geometric, sqrt_lambda_photometric, J_G, J_I, r_G,
                r_I);

        if (valid) {
            const scalar_t w_G = GetWeightFromRobustKernel(r_G);
            const scalar_t w_I = GetWeightFromRobustKernel(r_I);

            // Dump J, r into JtJ and Jtr
            int i = 0;
            for (int j = 0; j < 6; ++j) {
                for (int k = 0; k <= j; ++k) {
                    local_sum[i] +=
                            J_G[j] * w_G * J_G[k] + J_I[j] * w_I * J_I[k];
                    ++i;
                }
                local_sum[21 + j] += J_G[j] * w_G * r_G + J_I[j] * w_I * r_I;
            }
            local_sum[27] += r_G * r_G + r_I * r_I;
            local_sum[28] += 1;
        }
    }

    // Reduction.
    auto result = BlockReduce(temp_storage).Sum(local_sum);

    // Add result to global_sum.
    if (threadIdx.x == 0) {
#pragma unroll
        for (int i = 0; i < kReduceDim; ++i) {
            atomicAdd(&global_sum[i], result[i]);
        }
    }
}

void ComputePoseColoredICPCUDA(const core::Tensor &source_points,
                               const core::Tensor &source_colors,
                               const core::Tensor &target_points,
                               const core::Tensor &target_normals,
                               const core::Tensor &target_colors,
                               const core::Tensor &target_color_gradients,
                               const core::Tensor &correspondence_indices,
                               core::Tensor &pose,
                               float &residual,
                               int &inlier_count,
                               const core::Dtype &dtype,
                               const core::Device &device,
                               const registration::RobustKernel &kernel,
                               const double &lambda_geometric) {
    int n = source_points.GetLength();

    core::Tensor global_sum = core::Tensor::Zeros({29}, dtype, device);
    const dim3 blocks((n + kThread1DUnit - 1) / kThread1DUnit);
    const dim3 threads(kThread1DUnit);

    DISPATCH_FLOAT_DTYPE_TO_TEMPLATE(dtype, [&]() {
        scalar_t sqrt_lambda_geometric =
                static_cast<scalar_t>(sqrt(lambda_geometric));
        scalar_t sqrt_lambda_photometric =
                static_cast<scalar_t>(sqrt(1.0 - lambda_geometric));

        DISPATCH_ROBUST_KERNEL_FUNCTION(
                kernel.type_, scalar_t, kernel.scaling_parameter_,
                kernel.shape_parameter_, [&]() {
                    ComputePoseColoredICPKernelCUDA<<<
                            blocks, threads, 0, core::cuda::GetStream()>>>(
                            source_points.GetDataPtr<scalar_t>(),
                            source_colors.GetDataPtr<scalar_t>(),
                            target_points.GetDataPtr<scalar_t>(),
                            target_normals.GetDataPtr<scalar_t>(),
                            target_colors.GetDataPtr<scalar_t>(),
                            target_color_gradients.GetDataPtr<scalar_t>(),
                            correspondence_indices.GetDataPtr<int64_t>(),
                            sqrt_lambda_geometric, sqrt_lambda_photometric, n,
                            global_sum.GetDataPtr<scalar_t>(),
                            GetWeightFromRobustKernel);
                });
    });

    core::cuda::Synchronize();

    DecodeAndSolve6x6(global_sum, pose, residual, inlier_count);
}

template <typename scalar_t>
__global__ void ComputeInformationMatrixKernelCUDA(
        const scalar_t *target_points_ptr,
        const int64_t *correspondence_indices,
        const int n,
        scalar_t *global_sum) {
    // Reduce dimention for this function is 21
    typedef utility::MiniVec<scalar_t, 21> ReduceVec;
    // Create shared memory.
    typedef hipcub::BlockReduce<ReduceVec, kThread1DUnit> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    ReduceVec local_sum(static_cast<scalar_t>(0));

    const int workload_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (workload_idx < n) {
        scalar_t J_x[6] = {0}, J_y[6] = {0}, J_z[6] = {0};
        const bool valid = GetInformationJacobians<scalar_t>(
                workload_idx, target_points_ptr, correspondence_indices, J_x,
                J_y, J_z);

        if (valid) {
            int i = 0;
            for (int j = 0; j < 6; ++j) {
                for (int k = 0; k <= j; ++k) {
                    local_sum[i] +=
                            J_x[j] * J_x[k] + J_y[j] * J_y[k] + J_z[j] * J_z[k];
                    ++i;
                }
            }
        }
    }

    // Reduction.
    auto result = BlockReduce(temp_storage).Sum(local_sum);

    // Add result to global_sum.
    if (threadIdx.x == 0) {
#pragma unroll
        for (int i = 0; i < 21; ++i) {
            atomicAdd(&global_sum[i], result[i]);
        }
    }
}

void ComputeInformationMatrixCUDA(const core::Tensor &target_points,
                                  const core::Tensor &correspondence_indices,
                                  core::Tensor &information_matrix,
                                  const core::Dtype &dtype,
                                  const core::Device &device) {
    int n = correspondence_indices.GetLength();

    core::Tensor global_sum = core::Tensor::Zeros({21}, dtype, device);
    const dim3 blocks((n + kThread1DUnit - 1) / kThread1DUnit);
    const dim3 threads(kThread1DUnit);

    DISPATCH_FLOAT_DTYPE_TO_TEMPLATE(dtype, [&]() {
        scalar_t *global_sum_ptr = global_sum.GetDataPtr<scalar_t>();

        ComputeInformationMatrixKernelCUDA<<<blocks, threads, 0,
                                             core::cuda::GetStream()>>>(
                target_points.GetDataPtr<scalar_t>(),
                correspondence_indices.GetDataPtr<int64_t>(), n,
                global_sum_ptr);

        core::cuda::Synchronize();

        core::Tensor global_sum_cpu =
                global_sum.To(core::Device("CPU:0"), core::Float64);
        double *sum_ptr = global_sum_cpu.GetDataPtr<double>();

        // Information matrix is on CPU of type Float64.
        double *GTG_ptr = information_matrix.GetDataPtr<double>();

        int i = 0;
        for (int j = 0; j < 6; j++) {
            for (int k = 0; k <= j; k++) {
                GTG_ptr[j * 6 + k] = GTG_ptr[k * 6 + j] = sum_ptr[i];
                ++i;
            }
        }
    });
}

}  // namespace kernel
}  // namespace pipelines
}  // namespace t
}  // namespace open3d
